#include "hip/hip_runtime.h"
#include <generic/kernels.hpp>

#ifdef __HIPCC__

namespace Ensign {

template<class T>
__global__ void copy_R(int m, int n, T* Q, T* R, T w) {
  int i = threadIdx.x;
  int j = blockIdx.x;

  if(j>=i)
    R[i + j*n] = Q[i + j*m] * sqrt(w);
  else
    R[i + j*n] = 0.0;
}
template __global__ void copy_R(int, int, double*, double*, double);
template __global__ void copy_R(int, int, float*, float*, float);

template<class T>
__global__ void div_Q(int m, int n, T* Q, T w) {
  int i = threadIdx.x;
  int j = blockIdx.x;

  Q[i + j*m] /= sqrt(w);
}
template __global__ void div_Q(int, int, double*, double);
template __global__ void div_Q(int, int, float*, float);

template<class T>
__global__ void fill_gpu(int n, T* v, T alpha){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    v[idx] = alpha;
    idx += blockDim.x * gridDim.x;
  }
}
template __global__ void fill_gpu(int n, double*, double);
template __global__ void fill_gpu(int n, float*, float);

template<class T>
__global__ void ptw_mult_scal(int n, T* A, T alpha){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] *= alpha;
    idx += blockDim.x * gridDim.x;
  }
}
template __global__ void ptw_mult_scal(int, double*, double);
template __global__ void ptw_mult_scal(int, float*, float);

__global__ void ptw_mult_scal_cplx(int n, hipDoubleComplex* A, hipDoubleComplex alphac){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] = hipCmul(alphac,A[idx]);
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void ptw_mult_cplx(int n, hipDoubleComplex* A, double alpha){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  hipDoubleComplex alphac = make_hipDoubleComplex(alpha, 0.0);

  while(idx < n){
    A[idx] = hipCmul(alphac,A[idx]);
    idx += blockDim.x * gridDim.x;
  }
}



template<class T>
__global__ void ptw_mult_row_k(int nm, int n, T* A, T* v, T* B){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < nm){
    B[idx] = A[idx] * v[idx % n];
    idx += blockDim.x * gridDim.x;
  }
}
template __global__ void ptw_mult_row_k(int, int, double*, double*, double*);
template __global__ void ptw_mult_row_k(int, int, float*, float*, float*);


__global__ void ptw_mult_row_k(int nm, int n, hipDoubleComplex* A, hipDoubleComplex* v, hipDoubleComplex* B){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < nm){
    B[idx] = hipCmul(A[idx], v[idx % n]);
    idx += blockDim.x * gridDim.x;
  }
}

template<class T>
__global__ void ptw_sum(int n, T* A, T* B){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] += B[idx];
    idx += blockDim.x * gridDim.x;
  }
}
template __global__ void ptw_sum(int, double*, double*);
template __global__ void ptw_sum(int, float*, float*);


template<class T>
__global__ void ptw_sum_scal(int n, T* A, T alpha){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] += alpha;
    idx += blockDim.x * gridDim.x;
  }
}
template __global__ void ptw_sum_scal(int, double*, double);
template __global__ void ptw_sum_scal(int, float*, float);

__global__ void der_fourier(int n, hipDoubleComplex* A, double ax, double bx, int nx){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if(idx == 0){
    A[idx] = make_hipDoubleComplex(0.0, 0.0);
    idx += blockDim.x * gridDim.x;
  }
  while(idx < n){
    hipDoubleComplex c = make_hipDoubleComplex(0.0, (2.0*M_PI/(bx-ax))*idx*nx);
    A[idx] = hipCdiv(A[idx],c);
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void ptw_mult_row_cplx_fourier(int nm, int n, int nx, hipDoubleComplex* A, double ax, double bx){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < nm){
    hipDoubleComplex c = make_hipDoubleComplex(0.0, 2.0*M_PI/(nx*(bx-ax))*(idx%n));
    A[idx] = hipCmul(A[idx],c);
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void cplx_conv(int n, double* A, hipDoubleComplex* B){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    B[idx] = make_hipDoubleComplex(A[idx], 0.0);
    idx += blockDim.x * gridDim.x;
  }
}

__device__ hipDoubleComplex expim(double z){
    hipDoubleComplex out;
    sincos(z, &out.y, &out.x);
    return out;
}

__device__ hipDoubleComplex phi1im(double z){

    hipDoubleComplex out;

    if(abs(z) < 1e-7){
      out.x = 1.0 + z;
      out.y = z;
    }else{
      out.x = sin(z)/z;
      out.y = 2*(pow(sin(z/2.0),2))/z;
    }

    return out;

}

__device__ hipDoubleComplex phi2im(double z){

    hipDoubleComplex out;

    if(abs(z) < 1e-7){
      out.x = 0.5 + z;
      out.y = z;
    }else{
      out.x = 2*(pow(sin(z/2.0),2))/pow(z,2);
      out.y = (z-sin(z))/pow(z,2);
    }

    return out;

}


__global__ void exp_euler_fourier(int nm, int n, hipDoubleComplex* A, double* dc_r, double ts, hipDoubleComplex* T, double ax, double bx){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex tt = make_hipDoubleComplex(ts, 0.0);

  while(idx < nm){
    A[idx] = hipCmul(A[idx],expim(-ts*2.0*M_PI/(bx-ax)*(idx%n)*dc_r[idx / n]));
    A[idx] = hipCadd(A[idx],hipCmul(tt,hipCmul(phi1im(-ts*2.0*M_PI/(bx-ax)*(idx%n)*dc_r[idx / n]),T[idx])));
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void second_ord_stage_fourier(int nm, int n, hipDoubleComplex* A, double* dc_r, double ts, hipDoubleComplex* T, hipDoubleComplex* U, double ax, double bx){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex tt = make_hipDoubleComplex(ts, 0.0);

  while(idx < nm){
    A[idx] = hipCadd(A[idx],hipCmul(tt,hipCmul(phi2im(-ts*2.0*M_PI/(bx-ax)*(idx%n)*dc_r[idx / n]),hipCsub(U[idx],T[idx]))));
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void ptw_mult_scal(int n, double* A, double alpha, double* B){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    B[idx] = alpha*A[idx];
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void dmaxpy(int n, double* a, double* x, double* y){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    y[idx] = -(*a)*x[idx] + y[idx];
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void scale_unique(double* x, double alpha){
  *x *= alpha;
}

__global__ void scale_sqrt_unique(double* x, double alpha){
  *x = sqrt(*x * alpha);
}

__global__ void ptw_div_gs(int n, double* A, double* alpha){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] /= (*alpha);
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void ptw_mult(int n, double* A, double* B, double* C){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    C[idx] = A[idx] * B[idx];
    idx += blockDim.x * gridDim.x;
  }
}


__global__ void expl_euler(int n, double* A, double t, double* M1, double* M2){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] += t*(M1[idx]-M2[idx]);
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void rk4(int n, double* A, double t, double* M1, double* M2, double* M3){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    M2[idx]-= M3[idx];
    A[idx] = M1[idx] + t * M2[idx];
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void rk4_finalcomb(int n, double* A, double t, double* M1, double* M2, double* M3, double* M4, double* M5){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] += ((t/6.0) * (M1[idx]+2.0*M2[idx]+2.0*M3[idx]+M4[idx]-M5[idx]));
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void transpose_inplace(int n, double* A){

  int i = blockIdx.x % n ; // n number of rows
  int j = blockIdx.x / n;
  double tmp;
  if((i < n) && (j < i)){
    tmp = A[i+j*n];
    A[i+j*n] = A[j+i*n];
    A[j+i*n] = tmp;
  }
}

__global__ void der_fourier_2d(int N, int nx, int ny, hipDoubleComplex* A, double* lims, double nxx, hipDoubleComplex* B,hipDoubleComplex* C){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < N){
    int i = idx % nx;
    int j = idx / nx;

    if(j==ny/2){
      j = 0;
    }else if(j > ny/2){
      j -= ny;
    }

    hipDoubleComplex c1 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[1]-lims[0]))*i);
    hipDoubleComplex c2 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[3]-lims[2]))*j);

    hipDoubleComplex mm = hipCdiv(hipCadd(hipCmul(c1,c1),hipCmul(c2,c2)),make_hipDoubleComplex(nxx,0.0));

    if((i == 0) && (j == 0)){
      B[idx] = make_hipDoubleComplex(0.0, 0.0);
      C[idx] = make_hipDoubleComplex(0.0, 0.0);
    }else{
      B[idx] = hipCmul(A[idx],hipCdiv(c1,mm));
      C[idx] = hipCmul(A[idx],hipCdiv(c2,mm));
    }
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void ptw_mult_row_cplx_fourier_2d(int N, int nx, int ny, hipDoubleComplex* A, double* lims, double nxx, hipDoubleComplex* B, hipDoubleComplex* C){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < N){
    int i = (idx % (nx*ny)) % nx;
    int j = (idx % (nx*ny)) / nx;

    if(j==ny/2){
      j = 0;
    }else if(j > ny/2){
      j -= ny;
    }

    hipDoubleComplex c1 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[1]-lims[0]))*i*nxx);
    hipDoubleComplex c2 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[3]-lims[2]))*j*nxx);

    B[idx] = hipCmul(A[idx],c1);
    C[idx] = hipCmul(A[idx],c2);

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void exact_sol_exp_2d(int N, int nx, int ny, hipDoubleComplex* A, double* dc_r, double ts, double* lims, double nxx){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex nxx_c = make_hipDoubleComplex(nxx, 0.0);

  while(idx < N){
    int i = (idx % (nx*ny)) % nx;

    A[idx] = hipCmul(A[idx],hipCmul(expim(-ts*2.0*M_PI/(lims[1]-lims[0])*i*dc_r[idx / (nx*ny)]),nxx_c));

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void ptw_sum_complex(int n, hipDoubleComplex* A, hipDoubleComplex* B){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] = hipCadd(A[idx],B[idx]);
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void ptw_sum_3mat(int n, double* A, double* B, double* C){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] += B[idx];
    A[idx] += C[idx];
    idx += blockDim.x * gridDim.x;
  }
}
/*
__global__ void ptw_diff(int n, double* A, double* B){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] -= B[idx];
    idx += blockDim.x * gridDim.x;
  }
}
*/
template<class T>
__global__ void ptw_diff(int n, T* A, T* B){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < n){
    A[idx] -= B[idx];
    idx += blockDim.x * gridDim.x;
  }
}
template __global__ void ptw_diff(int, double*, double*);
template __global__ void ptw_diff(int, float*, float*);

__global__ void exp_euler_fourier_2d(int N, int nx, int ny, hipDoubleComplex* A, double* dc_r, double ts, double* lims, hipDoubleComplex* T){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex tt = make_hipDoubleComplex(ts, 0.0);

  while(idx < N){
    int j = (idx % (nx*ny)) / nx;

    if(j==ny/2){
      j = 0;
    }else if(j > ny/2){
      j -= ny;
    }

    A[idx] = hipCmul(A[idx],expim(-ts*(2.0*M_PI/(lims[3]-lims[2]))*j*dc_r[idx / (nx*ny)]));
    A[idx] = hipCadd(A[idx],hipCmul(tt,hipCmul(phi1im(-ts*(2.0*M_PI/(lims[3]-lims[2]))*j*dc_r[idx / (nx*ny)]),T[idx])));

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void second_ord_stage_fourier_2d(int N, int nx, int ny, hipDoubleComplex* A, double* dc_r, double ts, double* lims, hipDoubleComplex* T, hipDoubleComplex* U){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex tt = make_hipDoubleComplex(ts, 0.0);

  while(idx < N){
    int j = (idx % (nx*ny)) / nx;

    if(j==ny/2){
      j = 0;
    }else if(j > ny/2){
      j -= ny;
    }

    A[idx] = hipCadd(A[idx],hipCmul(tt,hipCmul(phi2im(-ts*(2.0*M_PI/(lims[3]-lims[2]))*j*dc_r[idx / (nx*ny)]),hipCsub(U[idx],T[idx]))));

    idx += blockDim.x * gridDim.x;
  }

}


__global__ void der_fourier_3d(int N, int nx, int ny, int nz, hipDoubleComplex* A, double* lims, double nxx, hipDoubleComplex* B, hipDoubleComplex* C, hipDoubleComplex* D){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < N){
    int i = idx % nx;
    int j = (idx / nx) % ny;
    int k = idx / (nx*ny);

    if(j==ny/2){
      j = 0;
    }else if(j > ny/2){
      j -= ny;
    }

    if(k==nz/2){
      k = 0;
    }else if(k > nz/2){
      k -= nz;
    }

    hipDoubleComplex c1 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[1]-lims[0]))*i);
    hipDoubleComplex c2 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[3]-lims[2]))*j);
    hipDoubleComplex c3 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[5]-lims[4]))*k);

    hipDoubleComplex mm = hipCdiv(hipCadd(hipCadd(hipCmul(c1,c1),hipCmul(c2,c2)),hipCmul(c3,c3)),make_hipDoubleComplex(nxx,0.0));

    if((i == 0) && (j == 0) && (k == 0)){
      B[idx] = make_hipDoubleComplex(0.0, 0.0);
      C[idx] = make_hipDoubleComplex(0.0, 0.0);
      D[idx] = make_hipDoubleComplex(0.0, 0.0);
    }else{
      B[idx] = hipCmul(A[idx],hipCdiv(c1,mm));
      C[idx] = hipCmul(A[idx],hipCdiv(c2,mm));
      D[idx] = hipCmul(A[idx],hipCdiv(c3,mm));
    }
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void ptw_mult_row_cplx_fourier_3d(int N, int nx, int ny, int nz, hipDoubleComplex* A, double* lims, double nxx, hipDoubleComplex* B, hipDoubleComplex* C, hipDoubleComplex* D){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < N){
    int i = (idx % (nx*ny*nz)) % nx;
    int j = ((idx % (nx*ny*nz)) / nx) % ny;
    int k = (idx % (nx*ny*nz)) / (nx*ny);

    if(j==ny/2){
      j = 0;
    }else if(j > ny/2){
      j -= ny;
    }

    if(k==nz/2){
      k = 0;
    }else if(k > nz/2){
      k -= nz;
    }

    hipDoubleComplex c1 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[1]-lims[0]))*i*nxx);
    hipDoubleComplex c2 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[3]-lims[2]))*j*nxx);
    hipDoubleComplex c3 = make_hipDoubleComplex(0.0, (2.0*M_PI/(lims[5]-lims[4]))*k*nxx);

    B[idx] = hipCmul(A[idx],c1);
    C[idx] = hipCmul(A[idx],c2);
    D[idx] = hipCmul(A[idx],c3);

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void exact_sol_exp_3d_a(int N, int nx, int ny, int nz, hipDoubleComplex* A, double* dc_r, double ts, double* lims){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < N){
    int i = (idx % (nx*ny*nz)) % nx;

    A[idx] = hipCmul(A[idx],expim(-ts*2.0*M_PI/(lims[1]-lims[0])*i*dc_r[idx / (nx*ny*nz)]));

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void exact_sol_exp_3d_b(int N, int nx, int ny, int nz, hipDoubleComplex* A, double* dc_r, double ts, double* lims, double nxx){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex nxx_c = make_hipDoubleComplex(nxx, 0.0);

  while(idx < N){
    int j = ((idx % (nx*ny*nz)) / nx) % ny;

    if(j==ny/2){
      j = 0;
    }else if(j > ny/2){
      j -= ny;
    }


    A[idx] = hipCmul(A[idx],hipCmul(expim(-ts*2.0*M_PI/(lims[3]-lims[2])*j*dc_r[idx / (nx*ny*nz)]),nxx_c));

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void exact_sol_exp_3d_c(int N, int nx, int ny, int nz, hipDoubleComplex* A, double* dc_r, double ts, double* lims){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while(idx < N){
    int j = ((idx % (nx*ny*nz)) / nx) % ny;

    if(j==ny/2){
      j = 0;
    }else if(j > ny/2){
      j -= ny;
    }


    A[idx] = hipCmul(A[idx],expim(-ts*2.0*M_PI/(lims[3]-lims[2])*j*dc_r[idx / (nx*ny*nz)]));

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void exact_sol_exp_3d_d(int N, int nx, int ny, int nz, hipDoubleComplex* A, double* dc_r, double ts, double* lims, double nxx){ // Very similar, maybe can be put together
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex nxx_c = make_hipDoubleComplex(nxx, 0.0);


  while(idx < N){
    int i = (idx % (nx*ny*nz)) % nx;

    A[idx] = hipCmul(A[idx],hipCmul(expim(-ts*2.0*M_PI/(lims[1]-lims[0])*i*dc_r[idx / (nx*ny*nz)]),nxx_c));

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void exp_euler_fourier_3d(int N, int nx, int ny, int nz, hipDoubleComplex* A, double* dc_r, double ts, double* lims, hipDoubleComplex* T){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex tt = make_hipDoubleComplex(ts, 0.0);

  while(idx < N){
    int k = (idx % (nx*ny*nz)) / (nx*ny);

    if(k > nz/2)
      k = k-nz;
    else if(k == nz/2)
      k = 0;

    A[idx] = hipCmul(A[idx],expim(-ts*(2.0*M_PI/(lims[5]-lims[4]))*k*dc_r[idx / (nx*ny*nz)]));
    A[idx] = hipCadd(A[idx],hipCmul(tt,hipCmul(phi1im(-ts*(2.0*M_PI/(lims[5]-lims[4]))*k*dc_r[idx / (nx*ny*nz)]),T[idx])));

    idx += blockDim.x * gridDim.x;
  }

}

__global__ void second_ord_stage_fourier_3d(int N, int nx, int ny, int nz, hipDoubleComplex* A, double* dc_r, double ts, double* lims, hipDoubleComplex* T, hipDoubleComplex* U){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  hipDoubleComplex tt = make_hipDoubleComplex(ts, 0.0);

  while(idx < N){
    int k = (idx % (nx*ny*nz)) / (nx*ny);

    if(k==nz/2){
      k = 0;
    }else if(k > nz/2){
      k -= nz;
    }

    A[idx] = hipCadd(A[idx],hipCmul(tt,hipCmul(phi2im(-ts*(2.0*M_PI/(lims[5]-lims[4]))*k*dc_r[idx / (nx*ny*nz)]),hipCsub(U[idx],T[idx]))));

    idx += blockDim.x * gridDim.x;
  }

}

} // namespace Ensign

#endif
